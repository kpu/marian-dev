#include "cuda_helpers.h"
#include "swap.h"
void copyCpuToGpu(const char * in, char * gpuOut);
void copyGpuToGpu(const char * in, char * gpuOut);

namespace marian {
    namespace swapper {
        void copyCpuToGpu(char * gpuOut, const char * in, size_t count, const marian::DeviceId& deviceId) {
            CUDA_CHECK(hipSetDevice(deviceId.no));
            CUDA_CHECK(hipMemcpy(gpuOut, in, count, hipMemcpyHostToDevice));
        }
        void copyGpuToGpu(char * gpuOut, const char * in, size_t count, const marian::DeviceId& deviceId) {
            CUDA_CHECK(hipSetDevice(deviceId.no));
            CUDA_CHECK(hipMemcpy(gpuOut, in, count, hipMemcpyDeviceToDevice));
        }
    }
}
